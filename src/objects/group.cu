#include "group.h"

Group::Group(int num_objects) : num(num_objects) {
    checkCudaErrors(hipMallocManaged(&objects, num * sizeof(Object3D *)));
    memset(objects, 0, num * sizeof(Object3D *));
}

Group::~Group() { checkCudaErrors(hipFree(objects)); }

__device__ bool Group::intersect(const Ray &ray, Hit &hit, float t_min, hiprandState *rand_state) {
    bool res = false;
    for (int i = 0; i < num; i++) {
        if (objects[i] != nullptr && objects[i]->intersect(ray, hit, t_min, rand_state)) {
            res = true;
        }
    }
    return res;
}

void Group::addObject(int index, Object3D *obj) { objects[index] = obj; }
