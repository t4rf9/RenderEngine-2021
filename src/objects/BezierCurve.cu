#include "BezierCurve.h"

BezierCurve::BezierCurve(const std::vector<Vector3f> &points) : Curve(points) {
    if (points.size() < 4 || points.size() % 3 != 1) {
        printf("Number of control points of a BezierCurve must be 3n+1!\n");
        exit(0);
    }

    checkCudaErrors(hipMallocManaged(&B, num_controls * sizeof(float *)));
    for (int i = 0; i < num_controls; i++) {
        checkCudaErrors(hipMallocManaged(&B[i], (i + 1) * sizeof(float)));
    }
}

BezierCurve::~BezierCurve() {
    for (int i = 0; i <= n; i++) {
        checkCudaErrors(hipFree(B[i]));
    }
    checkCudaErrors(hipFree(B));
}

__device__ CurvePoint BezierCurve::curve_point_at_t(float t) {
    // calculate B[p][q] = B_{q, p}(t)
    B[0][0] = 1.f;
    for (int p = 1; p <= n; p++) {
        B[p][0] = (1 - t) * B[p - 1][0];
        for (int q = 1; q < p; q++) {
            B[p][q] = t * B[p - 1][q - 1] + (1 - t) * B[p - 1][q];
        }
        B[p][p] = t * B[p - 1][p - 1];
    }

    Vector3f V = Vector3f(0.f);
    for (int j = 0; j <= n; j++) {
        V += B[n][j] * controls[j];
    }

    Vector3f T = Vector3f(0.f);
    for (int j = 0; j < n; j++) {
        T += B[n - 1][j] * (controls[j + 1] - controls[j]);
    }
    T *= n;

    return {V, T};
}

/*
__device__ void BezierCurve::discretize(int resolution, std::vector<CurvePoint> &data) {
    data.clear();
    // PA3: fill in data vector

    double step = 1. / resolution;
    for (int i = 0; i <= resolution; i++) {
        double t = i * step;
        data.push_back(curve_point_at_t(t));
    }
}
*/
