#include "objects/BoundingObject.h"

void *BoundingObject::operator new(std::size_t sz) {
    void *res;
    checkCudaErrors(hipMallocManaged(&res, sz));
    return res;
}

void *BoundingObject::operator new[](std::size_t sz) {
    void *res;
    checkCudaErrors(hipMallocManaged(&res, sz));
    return res;
}

void BoundingObject::operator delete(void *ptr) { checkCudaErrors(hipFree(ptr)); }

void BoundingObject::operator delete[](void *ptr) { checkCudaErrors(hipFree(ptr)); }
