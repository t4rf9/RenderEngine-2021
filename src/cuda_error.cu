#include "cuda_error.h"

void check_cuda(hipError_t result, char const *const func, const char *const file,
                int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":"
                  << line << " '" << func << "' \n";
        std::cerr << hipGetErrorName(result) << ":\t" << hipGetErrorString(result) << std::endl;
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}