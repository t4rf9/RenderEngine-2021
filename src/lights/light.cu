#include "lights/light.h"

void *Light::operator new(std::size_t sz) {
    void *res;
    checkCudaErrors(hipMallocManaged(&res, sz));
    return res;
}

void *Light::operator new[](std::size_t sz) {
    void *res;
    checkCudaErrors(hipMallocManaged(&res, sz));
    return res;
}

void Light::operator delete(void *ptr) { checkCudaErrors(hipFree(ptr)); }

void Light::operator delete[](void *ptr) { checkCudaErrors(hipFree(ptr)); }
