#include "hip/hip_runtime.h"
#include "Vector2f.h"

#include <cassert>
#include <cmath>
#include <cstdio>
#include <cstdlib>

#include "Vector3f.h"

//////////////////////////////////////////////////////////////////////////
// Public
//////////////////////////////////////////////////////////////////////////

// static
const Vector2f Vector2f::ZERO = Vector2f(0, 0);

// static
const Vector2f Vector2f::UP = Vector2f(0, 1);

// static
const Vector2f Vector2f::RIGHT = Vector2f(1, 0);

__host__ __device__ Vector2f::Vector2f(float f) {
    m_elements[0] = f;
    m_elements[1] = f;
}

__host__ __device__ Vector2f::Vector2f(float x, float y) {
    m_elements[0] = x;
    m_elements[1] = y;
}

__host__ __device__ Vector2f::Vector2f(const Vector2f &rv) {
    m_elements[0] = rv[0];
    m_elements[1] = rv[1];
}

__host__ __device__ Vector2f &Vector2f::operator=(const Vector2f &rv) {
    if (this != &rv) {
        m_elements[0] = rv[0];
        m_elements[1] = rv[1];
    }
    return *this;
}

__host__ __device__ const float &Vector2f::operator[](int i) const { return m_elements[i]; }

__host__ __device__ float &Vector2f::operator[](int i) { return m_elements[i]; }

__host__ __device__ float &Vector2f::x() { return m_elements[0]; }

__host__ __device__ float &Vector2f::y() { return m_elements[1]; }

__host__ __device__ float Vector2f::x() const { return m_elements[0]; }

__host__ __device__ float Vector2f::y() const { return m_elements[1]; }

__host__ __device__ Vector2f Vector2f::xy() const { return *this; }

__host__ __device__ Vector2f Vector2f::yx() const { return Vector2f(m_elements[1], m_elements[0]); }

__host__ __device__ Vector2f Vector2f::xx() const { return Vector2f(m_elements[0], m_elements[0]); }

__host__ __device__ Vector2f Vector2f::yy() const { return Vector2f(m_elements[1], m_elements[1]); }

__host__ __device__ Vector2f Vector2f::normal() const {
    return Vector2f(-m_elements[1], m_elements[0]);
}

__host__ __device__ float Vector2f::abs() const { return sqrt(absSquared()); }

__host__ __device__ float Vector2f::absSquared() const {
    return m_elements[0] * m_elements[0] + m_elements[1] * m_elements[1];
}

__host__ __device__ void Vector2f::normalize() {
    float norm = abs();
    m_elements[0] /= norm;
    m_elements[1] /= norm;
}

__host__ __device__ Vector2f Vector2f::normalized() const {
    float norm = abs();
    return Vector2f(m_elements[0] / norm, m_elements[1] / norm);
}

__host__ __device__ void Vector2f::negate() {
    m_elements[0] = -m_elements[0];
    m_elements[1] = -m_elements[1];
}

Vector2f::operator const float *() const { return m_elements; }

Vector2f::operator float *() { return m_elements; }

void Vector2f::print() const { printf("< %.4f, %.4f >\n", m_elements[0], m_elements[1]); }

__host__ __device__ Vector2f &Vector2f::operator+=(const Vector2f &v) {
    m_elements[0] += v.m_elements[0];
    m_elements[1] += v.m_elements[1];
    return *this;
}

__host__ __device__ Vector2f &Vector2f::operator-=(const Vector2f &v) {
    m_elements[0] -= v.m_elements[0];
    m_elements[1] -= v.m_elements[1];
    return *this;
}

__host__ __device__ Vector2f &Vector2f::operator*=(float f) {
    m_elements[0] *= f;
    m_elements[1] *= f;
    return *this;
}

// static
__host__ __device__ float Vector2f::dot(const Vector2f &v0, const Vector2f &v1) {
    return v0[0] * v1[0] + v0[1] * v1[1];
}

// static
__host__ __device__ Vector3f Vector2f::cross(const Vector2f &v0, const Vector2f &v1) {
    return Vector3f(0, 0, v0.x() * v1.y() - v0.y() * v1.x());
}

// static
__host__ __device__ Vector2f Vector2f::lerp(const Vector2f &v0, const Vector2f &v1, float alpha) {
    return alpha * (v1 - v0) + v0;
}

//////////////////////////////////////////////////////////////////////////
// Operator overloading
//////////////////////////////////////////////////////////////////////////

__host__ __device__ Vector2f operator+(const Vector2f &v0, const Vector2f &v1) {
    return Vector2f(v0.x() + v1.x(), v0.y() + v1.y());
}

__host__ __device__ Vector2f operator-(const Vector2f &v0, const Vector2f &v1) {
    return Vector2f(v0.x() - v1.x(), v0.y() - v1.y());
}

__host__ __device__ Vector2f operator*(const Vector2f &v0, const Vector2f &v1) {
    return Vector2f(v0.x() * v1.x(), v0.y() * v1.y());
}

__host__ __device__ Vector2f operator/(const Vector2f &v0, const Vector2f &v1) {
    return Vector2f(v0.x() / v1.x(), v0.y() / v1.y());
}

__host__ __device__ Vector2f operator-(const Vector2f &v) { return Vector2f(-v.x(), -v.y()); }

__host__ __device__ Vector2f operator*(float f, const Vector2f &v) {
    return Vector2f(f * v.x(), f * v.y());
}

__host__ __device__ Vector2f operator*(const Vector2f &v, float f) {
    return Vector2f(f * v.x(), f * v.y());
}

__host__ __device__ Vector2f operator/(const Vector2f &v, float f) {
    return Vector2f(v.x() / f, v.y() / f);
}

__host__ __device__ bool operator==(const Vector2f &v0, const Vector2f &v1) {
    return (v0.x() == v1.x() && v0.y() == v1.y());
}

__host__ __device__ bool operator!=(const Vector2f &v0, const Vector2f &v1) { return !(v0 == v1); }
